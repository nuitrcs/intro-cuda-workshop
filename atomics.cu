
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 1000000
#define ARRAY_SIZE  16 

#define BLOCK_WIDTH 1000 

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
    // which thread is this?
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // each thread to increment consecutive elements, wrapping at ARRAY_SIZE
    i = i % ARRAY_SIZE;
    g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
    // which thread is this?
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // each thread to increment consecutive elements, wrapping at ARRAY_SIZE
    i = i % ARRAY_SIZE;
    atomicAdd(&g[i], 1);

}

int main(int argc,char **argv)
{
    // Informative printout
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);

    // Fill the device memory area with zeros  .
    hipMemset((void *) d_array, 0, ARRAY_BYTES);

    // Call kernel with atomic operation
    increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);

    // Call kernel without atomic operation
//    increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);

    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);

    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}
